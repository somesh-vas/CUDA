#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(int *a, int *b, int *c, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    const int size = 1000000;
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    int *hostA, *hostB, *hostC;
    int *deviceA, *deviceB, *deviceC;

    // Allocate host memory
    hostA = new int[size];
    hostB = new int[size];
    hostC = new int[size];

    // Initialize host arrays
    for (int i = 0; i < size; ++i) {
        hostA[i] = i;
        hostB[i] = i * 2;
    }

    // Allocate device memory
    hipMalloc((void**)&deviceA, size * sizeof(int));
    hipMalloc((void**)&deviceB, size * sizeof(int));
    hipMalloc((void**)&deviceC, size * sizeof(int));

    // Copy data from host to device
    hipMemcpy(deviceA, hostA, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(deviceA, deviceB, deviceC, size);

    // Copy result from device to host
    hipMemcpy(hostC, deviceC, size * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    // Free host memory
    delete[] hostA;
    delete[] hostB;
    delete[] hostC;

    return 0;
}
