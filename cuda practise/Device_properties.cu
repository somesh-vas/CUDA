#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    hipDeviceProp_t prop;
    int device;

    // Get the device count
    hipGetDeviceCount(&device);

    if (device == 0) {
        std::cerr << "No CUDA-capable device found." << std::endl;
        return 1;
    }

    // Get device properties
    hipGetDeviceProperties(&prop, 0);  // Assuming you are using the first GPU

    // Print device information
    std::cout << "Device Name: " << prop.name << std::endl;
    std::cout << "Global Memory: " << prop.totalGlobalMem << " bytes" << std::endl;
    std::cout << "Shared Memory Per Block: " << prop.sharedMemPerBlock << " bytes" << std::endl;
    std::cout << "Constant Memory: " << prop.totalConstMem << " bytes" << std::endl;
    std::cout << "Texture Memory: " << prop.textureAlignment << " bytes" << std::endl;
    std::cout << "Local Memory Per Block: " << prop.localMemoryPerBlock << " bytes" << std::endl;

    return 0;
}
