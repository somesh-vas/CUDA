#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

#include <stdio.h>
#include <stdlib.h>

// CUDA kernel to compute the sum of an array
__global__ void sumArray(int* d_array, int* d_result, int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int step = gridDim.x * blockDim.x;

    int sum = 0;
    for (int i = tid; i < N; i += step) {
        sum += d_array[i];
    }
    d_result[tid] = sum;
}

int main() {
    int N = 1024; // Array size
    int* h_array, * d_array, * d_result;
    int result = 0;

    // Allocate host and device memory
    h_array = (int*)malloc(N * sizeof(int));
    hipMalloc((void**)&d_array, N * sizeof(int));
    hipMalloc((void**)&d_result, N * sizeof(int));

    // Initialize host array
    for (int i = 0; i < N; i++) {
        h_array[i] = i;
    }

    // Copy data from host to device
    hipMemcpy(d_array, h_array, N * sizeof(int), hipMemcpyHostToDevice);

    // Define thread block and grid dimensions
    dim3 blockDim(256);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x);

    // Launch the kernel
    sumArray << <gridDim, blockDim >> > (d_array, d_result, N);

    // Copy the result from device to host
    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("Sum: %d\n", result);

    // Cleanup
    free(h_array);
    hipFree(d_array);
    hipFree(d_result);

    return 0;
}
