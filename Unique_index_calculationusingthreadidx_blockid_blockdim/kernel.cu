#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void uniq(int * input)
{
    int tid = threadIdx.x;
    printf("threadIdx : %d, value : %d \n", tid, input[tid]);
}
__global__ void uniq_grid_1d(int* input) {
    int tid = threadIdx.x;
    int offset = blockIdx.x * blockDim.x;
    int gid = tid + offset;
    printf("blockIdx.x : %d, threadIdx.x : %d, grid : %d, value : %d \n",
        blockIdx.x, tid, gid, input[gid]);
}
__global__ void uniq_grid_2d(int* input) {
    int tid = threadIdx.x;
    int block_offset = blockIdx.x * blockDim.x;
    int row_offset = blockDim.x * gridDim.x * blockIdx.y;
    int gid = tid + block_offset + row_offset;
    printf("blockIdx.x : %d,blockIdx.y : %d, threadIdx.x : %d, grid : %d, value : %d \n",
        blockIdx.x,blockIdx.y, tid, gid, input[gid]);
}
__global__ void uniq_grid_2d_2d(int* input) {
    int tid = blockDim.x * threadIdx.y + threadIdx.x;
    int num_threads_in_a_block = blockDim.x * blockDim.y;
    int block_offset = blockIdx.x * num_threads_in_a_block;
    int num_threads_in_a_row = num_threads_in_a_block * gridDim.x;
    int row_offset = num_threads_in_a_row * blockIdx.y;
    int gid = tid + block_offset + row_offset;
    printf("blockIdx.x : %d,blockIdx.y : %d, threadIdx.x : %d, grid : %d, value : %d \n",
        blockIdx.x, blockIdx.y, tid, gid, input[gid]);
}


/*
* index =(blockIdx.X X blockDim.X) + threadIdx.X
* index = offset + tid
* 
* general index calculation
* index = row offset + block offset + tid
* index = (number of threads in one thread block row * blockIdx.y) + number of threads in * thread block * blockIdx.x + threadIdx.x
* 
* -- number of threads in one row = gridDim.x * blockDim.x
* -- number of threads in thread block = blockDim.x
*/
int main() {
    int array_size = 16;
    int array_byte_size = sizeof(int) * array_size;
    int h_data[16];

    for (int i = 0; i < array_size; i++) {
        h_data[i] = i + 1;
        printf("%d ", h_data[i]);
    }

    printf("\n \n");

    int* d_data;
    hipMalloc((void**)&d_data, array_byte_size);
    hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

    dim3 block(2,2);
    dim3 grid(2,2);

    uniq_grid_2d_2d<< < grid, block >> > (d_data);
    hipDeviceSynchronize();

    hipDeviceReset();
    return 0;
}