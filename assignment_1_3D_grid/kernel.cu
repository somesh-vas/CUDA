#include "hip/hip_runtime.h"
﻿#include <stdio.h>

__global__ void printThreadInfo() {
    int threadIdX = threadIdx.x;
    int threadIdY = threadIdx.y;
    int threadIdZ = threadIdx.z;

    int blockIdX = blockIdx.x;
    int blockIdY = blockIdx.y;
    int blockIdZ = blockIdx.z;

    int gridDimX = gridDim.x;
    int gridDimY = gridDim.y;
    int gridDimZ = gridDim.z;

    printf("ThreadIdx: (%d, %d, %d)\n", threadIdX, threadIdY, threadIdZ);
    printf("BlockIdx: (%d, %d, %d)\n", blockIdX, blockIdY, blockIdZ);
    printf("GridDim: (%d, %d, %d)\n", gridDimX, gridDimY, gridDimZ);
}

int main() {
    dim3 blockDimensions(2, 2, 2); // 2 threads in each dimension for the block
    dim3 gridDimensions(2, 2, 2);   // 4 threads in all dimensions for the grid

    printThreadInfo << <gridDimensions, blockDimensions >> > ();
    hipDeviceSynchronize(); // Wait for the GPU to finish

    return 0;
}
