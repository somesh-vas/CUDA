#include <stdio.h>
#include "decrypt.h"
#include "params.h"
#include <hip/hip_runtime.h>
typedef unsigned short uint16_t;
typedef uint16_t gf;


int decrypt(unsigned char *e, const unsigned char *sk, const unsigned char *c){
    gf g[ SYS_T+1 ]; // goppa polynomial
    gf L[ SYS_N ]; // support

    // Call the host function to compute g
    compute_g_host(g, sk);
    support_gen(L, sk);
    

return 1;
}
