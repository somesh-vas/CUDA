#include "hip/hip_runtime.h"
#include "decrypt.h"
#include "params.h"
#include <hip/hip_runtime.h>
#include <stdio.h>


typedef unsigned short uint16_t;
typedef uint16_t gf;



__device__ uint16_t load_gf(const unsigned char *src) {
    uint16_t a;

    a = src[1];
    a <<= 8;  // Left-shift by 8 bits (one byte)
    a |= src[0];

    return a & GFMASK;
}

__global__ void compute_g_cuda(uint16_t *g, const unsigned char *sk) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < SYS_T) {
        g[tid] = load_gf(sk + 2 * tid);
    }
}
void compute_g_host(uint16_t *h_g, const unsigned char *h_sk) {
    // Allocate memory on the GPU
    uint16_t *d_g;
    unsigned char *d_sk;
    hipMalloc((void**)&d_g, SYS_T * sizeof(uint16_t));
    hipMalloc((void**)&d_sk, 2 * SYS_T * sizeof(unsigned char));

    // Copy data from host to device
    hipMemcpy(d_sk, h_sk, 2 * SYS_T * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 blockDim(256);
    dim3 gridDim((SYS_T + blockDim.x - 1) / blockDim.x);

    // Launch the CUDA kernel
    compute_g_cuda<<<gridDim, blockDim>>>(d_g, d_sk);

    // Synchronize to wait for the kernel to finish
    hipDeviceSynchronize();

    // Copy the result from device to host
    hipMemcpy(h_g, d_g, SYS_T * sizeof(uint16_t), hipMemcpyDeviceToHost);

    // Print the computed values of g
    printf("Computed g: ");
    for (int i = 0; i < SYS_T; ++i) {
        printf("%04X ", h_g[i]);
    }
    printf("\n");

    // Free allocated memory on the GPU
    hipFree(d_g);
    hipFree(d_sk);
}