#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void hello_cuda()
{
	printf("hello cuda\n");
}

int main() {
	dim3 block(4, 1, 1);
	dim3 grid(8, 1, 1);
	hello_cuda << < grid, block >> > ();
	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}

